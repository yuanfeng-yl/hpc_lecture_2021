
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
using namespace std;

__global__ void matrix(int N, float *A, float *B, float* C){
    int i = blockIdx.x / N;
    int j = blockIdx.x % N;
    int k = threadIdx.x;
    atomicAdd(C+N*i+j, A[N*i+k]*B[N*k+j]);
}

int main(int argc, char** argv) {
    const int N = 256;
    float *A;
    float *B;
    float *C;
    hipMallocManaged(&A, N*N*sizeof(float));
    hipMallocManaged(&B, N*N*sizeof(float));
    hipMallocManaged(&C, N*N*sizeof(float));
    
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
        }
    }
    
    auto tic = chrono::steady_clock::now();
    matrix<<<N*N, N>>>(N, A, B, C);
    hipDeviceSynchronize();
    auto toc = chrono::steady_clock::now();
    
    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            for (int k=0; k<N; k++)
                C[N*i+j] -= A[N*i+k] * B[N*k+j];
    double err = 0;
    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            err += fabs(C[N*i+j]);
    double time = chrono::duration<double>(toc-tic).count();
    printf("N    : %d\n",N);
    printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
    printf("error: %lf\n",err/N/N);
}